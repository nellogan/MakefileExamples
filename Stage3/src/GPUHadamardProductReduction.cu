#include "hip/hip_runtime.h"
#include "GPUHadamardProductReduction.h"

__global__ void GPUHadamardProduct(float* arr1, float* arr2)
{
    int global_thread_id    = threadIdx.x + blockIdx.x * blockDim.x;
    arr1[global_thread_id] *= arr2[global_thread_id];
}


float GPUHadamardProductReduction(int n)
{
    float* arr1;
    float* arr2;

    int block_size = 256;
    int num_blocks = (n + block_size - 1) / block_size;
    /*
    If padded_size is set to n: will trigger compute-sanitize to throw an error exit code
    (assuming (n % block_size != 0) from Test_GPUHadamardProductReduction).
    */
//     int padded_size =  n;
    int padded_size =  num_blocks*block_size;

    hipMallocManaged(&arr1, padded_size*sizeof(float));
    hipMallocManaged(&arr2, padded_size*sizeof(float));

    for ( int i=0; i<n; i++ )
    {
        arr1[i] = 3.0f;
        arr2[i] = 4.0f;
    }

    for ( int i=n; i<padded_size; i++ )
    {
        arr1[i] = 0.0f;
        arr2[i] = 0.0f;
    }

    GPUHadamardProduct<<<num_blocks, block_size>>>(arr1, arr2);
    hipDeviceSynchronize();

    float sum = 0.0f;
    for ( int i=0; i<n; i++ )
    {
        sum += arr1[i];
    }

    hipFree(arr1);
    hipFree(arr2);

    return sum;
}