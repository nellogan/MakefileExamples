#include "hip/hip_runtime.h"
#include "CUDANoUnitTest.h"

__global__ void CUDANoUnitTestKernel()
{
    int global_thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    printf("GPU Kernel in CUDANoUnitTest.cu says: Hello! from global_thread_id %d\n", global_thread_id);
}

int CUDANoUnitTest(int n)
{
    CUDANoUnitTestKernel<<<2, 2>>>();
    hipDeviceSynchronize();
    return n;
}